#include "hip/hip_runtime.h"
#include <iostream>
#include <CosineVector.cuh>
#include <ScalarMulRunner.cuh>

__global__
void ScalarMulBlock(int numElements, float* vector1, float* vector2, float* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
    
    result[blockIdx.x] = 0;
	for (int i = index; i < numElements; i += stride) {
		atomicAdd(&result[blockIdx.x], vector1[i] * vector2[i]);
	}
}

__global__ void Reduce(float* array, float* res) {
    extern __shared__ float shared[];

    int threadId = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    shared[threadId] = array[index];
    __syncthreads();
    
    for (int curSize = blockDim.x / 2; curSize > 0; curSize /= 2) {
        if (threadId < curSize) {
            shared[threadId] += shared[threadId + curSize];
        }
        __syncthreads();
    }

    if (threadId == 0) {
        res[blockIdx.x] = shared[0];
    }
}

float CosineVector(int numElements, float* vector1, float* vector2, int blockSize) {
    float *d_vector1, *d_vector2, *d_mul, *d_res;
    
    int nBlocks = (numElements + blockSize - 1) / blockSize;

    hipMalloc(&d_vector1, numElements * sizeof(float));
    hipMalloc(&d_vector2, numElements * sizeof(float));
    hipMalloc(&d_mul, nBlocks * blockSize * sizeof(float));
    hipMalloc(&d_res, sizeof(float));
    hipMemcpy(d_vector1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    ScalarMulBlock<<<nBlocks, blockSize>>>(numElements, d_vector1, d_vector2, d_mul);
    hipDeviceSynchronize();
    
    float nBlocks1 = (nBlocks + blockSize - 1) / blockSize;
    Reduce<<<nBlocks1, blockSize, blockSize * sizeof(float)>>>(d_mul, d_res);
    hipDeviceSynchronize();
    
    float scalMul = 0;
    hipMemcpy(&scalMul, d_res, sizeof(float), hipMemcpyDeviceToHost);
    
    ScalarMulBlock<<<nBlocks, blockSize>>>(numElements, d_vector1, d_vector1, d_mul);
    hipDeviceSynchronize();
    
    Reduce<<<nBlocks1, blockSize, blockSize * sizeof(float)>>>(d_mul, d_res);
    hipDeviceSynchronize();
    
    float len1sq = 0;
    hipMemcpy(&len1sq, d_res, sizeof(float), hipMemcpyDeviceToHost);
    
    ScalarMulBlock<<<nBlocks, blockSize>>>(numElements, d_vector2, d_vector2, d_mul);
    hipDeviceSynchronize();
    
    Reduce<<<nBlocks1, blockSize, blockSize * sizeof(float)>>>(d_mul, d_res);
    hipDeviceSynchronize();
    
    float len2sq = 0;
    hipMemcpy(&len2sq, d_res, sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float run_time_ms = 0;
    hipEventElapsedTime(&run_time_ms, start, stop);
    std::cout << run_time_ms << std::endl;

    hipFree(d_vector1);
    hipFree(d_vector2);
    hipFree(d_mul);
    hipFree(d_res);
    return scalMul / (sqrt(len1sq) * sqrt(len2sq));
}
#include "hip/hip_runtime.h"
#include <iostream>
#include <ScalarMul.cuh>
#include <ScalarMulRunner.cuh>

__global__ void KernelMul(int numElements, float* x, float* y, float* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < numElements; i += stride) {
		result[i] = x[i] * y[i];
	}
}

__global__ void Reduce(float* array, float* res) {
    extern __shared__ float shared[];

    int threadId = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
//printf("%d\n", index);
    shared[threadId] = array[index];
    __syncthreads();
    
    for (int curSize = 1; curSize < blockDim.x; curSize *= 2) {
        int i = 2 * curSize * threadId;

        if (i < blockDim.x) {
            shared[i] += shared[i + curSize];
        }
        __syncthreads();
    }

    if (threadId == 0) {
        res[blockIdx.x] = shared[0];
    }
}

float ScalarMulTwoReductions(int numElements, float* vector1, float* vector2, int blockSize) {
    float *d_vector1, *d_vector2, *d_mul, *d_reduce, *d_res;

    int nBlocks = (numElements + blockSize - 1) / blockSize;
    
    hipMalloc(&d_vector1, numElements * sizeof(float));
    hipMalloc(&d_vector2, numElements * sizeof(float));
    hipMalloc(&d_mul, numElements * sizeof(float));
    hipMalloc(&d_reduce, nBlocks * blockSize * sizeof(float));
    hipMalloc(&d_res, sizeof(float));

    hipMemcpy(d_vector1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    KernelMul<<<nBlocks, blockSize>>>(numElements, d_vector1, d_vector2, d_mul);
    hipDeviceSynchronize();
    
    Reduce<<<nBlocks, blockSize, blockSize * sizeof(float)>>>(d_mul, d_reduce);
    hipDeviceSynchronize();
    
    nBlocks = (nBlocks + blockSize - 1) / blockSize;
    Reduce<<<nBlocks, blockSize, blockSize * sizeof(float)>>>(d_reduce, d_res);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float run_time_ms = 0;
    hipEventElapsedTime(&run_time_ms, start, stop);
    std::cout << run_time_ms << std::endl;
    
    float res = 0;
    hipMemcpy(&res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_vector1);
    hipFree(d_vector2);
    hipFree(d_mul);
    hipFree(d_reduce);
    hipFree(d_res);
    return res;
}

float ScalarMulSumPlusReduction(int numElements, float* vector1, float* vector2, int blockSize) {
    float *d_vector1, *d_vector2, *d_mul, *d_res;
    
    int nBlocks = (numElements + blockSize - 1) / blockSize;

    hipMalloc(&d_vector1, numElements * sizeof(float));
    hipMalloc(&d_vector2, numElements * sizeof(float));
    hipMalloc(&d_mul, nBlocks * blockSize * sizeof(float));
    hipMalloc(&d_res, sizeof(float));
    hipMemcpy(d_vector1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    ScalarMulBlock<<<nBlocks, blockSize>>>(numElements, d_vector1, d_vector2, d_mul);
    hipDeviceSynchronize();
    
    nBlocks = (nBlocks + blockSize - 1) / blockSize;
    Reduce<<<nBlocks, blockSize, blockSize * sizeof(float)>>>(d_mul, d_res);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float run_time_ms = 0;
    hipEventElapsedTime(&run_time_ms, start, stop);
    std::cout << run_time_ms << std::endl;
    
    float res = 0;
    hipMemcpy(&res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_vector1);
    hipFree(d_vector2);
    hipFree(d_mul);
    hipFree(d_res);
    return res;
}


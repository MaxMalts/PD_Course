#include <iostream>
#include <cmath>
#include "KernelAdd.cuh"

int main() {
    int nElements = 0, blockSize = 0;
    std::cin >> nElements >> blockSize;
    
    float *x, *y, *res;
    
    hipMallocManaged(&x, nElements * sizeof(float));
    hipMallocManaged(&y, nElements * sizeof(float));
    hipMallocManaged(&res, nElements * sizeof(float));

    for (int i = 0; i < nElements; ++i) {
        x[i] = 2.0f;
        y[i] = 3.0f;
    }

    int nBlocks = (nElements + blockSize - 1) / blockSize;

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    KernelAdd<<<nBlocks, blockSize>>>(nElements, x, y, res);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipDeviceSynchronize();	
    
    float run_time_ms = 0;
    hipEventElapsedTime(&run_time_ms, start, stop);
    std::cout << run_time_ms << std::endl;

    hipFree(x);
    hipFree(y);
    hipFree(res);
    return 0;
}
#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "KernelMatrixAdd.cuh"

int main() {
    int nElements = 0, blockSize = 0;
    std::cin >> nElements >> blockSize;
    int size = static_cast<int>(sqrt(nElements));
    
    float* a = (float*)malloc(size * size * sizeof(float));
    float* b = (float*)malloc(size * size * sizeof(float));
    float* res = (float*)malloc(size * size * sizeof(float));
    
    float *d_a, *d_b, *d_res;
    size_t pitch = 0;
    hipMallocPitch(&d_a, &pitch, size * sizeof(float), size);
    hipMallocPitch(&d_b, &pitch, size * sizeof(float), size);
    hipMallocPitch(&d_res, &pitch, size * sizeof(float), size);

    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            a[i * size + j] = 2.0f;
            b[i * size + j] = 3.0f;
        }
    }
    
    hipMemcpy2D(d_a, pitch, a, size * sizeof(float), size * sizeof(float), size, hipMemcpyHostToDevice);
    hipMemcpy2D(d_b, pitch, b, size * sizeof(float), size * sizeof(float), size, hipMemcpyHostToDevice);

    int nBlocks = (size * pitch + blockSize - 1) / blockSize;

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    KernelMatrixAdd<<<nBlocks, blockSize>>>(size, size, pitch, d_a, d_b, d_res);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipDeviceSynchronize();
    
    float run_time_ms = 0;
    hipEventElapsedTime(&run_time_ms, start, stop);
    std::cout << run_time_ms << std::endl;
    
    hipMemcpy2D(res, size * sizeof(float), d_res, pitch, size * sizeof(float), size, hipMemcpyDeviceToHost);
    
    free(a);
    free(b);
    free(res);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
    return 0;
}
#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "MatrixVectorMul.cuh"

int main() {
    int nElements = 0, blockSize = 0;
    std::cin >> nElements >> blockSize;
    int size = static_cast<int>(sqrt(nElements));
    
    float* a = (float*)malloc(size * size * sizeof(float));
    float* v = (float*)malloc(size * sizeof(float));
    float* res = (float*)malloc(size * sizeof(float));
    
    float *d_a, *d_v, *d_res;
    size_t pitch = 0;
    hipMallocPitch(&d_a, &pitch, size * sizeof(float), size);
    hipMalloc(&d_v, size * sizeof(float));
    hipMalloc(&d_res, size * sizeof(float));

    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            a[i * size + j] = 2.0f;
        }
    }
    for (int i = 0; i < size; ++i) {
        v[i] = 3.0f;
    }
    
    hipMemcpy2D(d_a, pitch, a, size * sizeof(float), size * sizeof(float), size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, size * sizeof(float), hipMemcpyHostToDevice);

    int nBlocks = (size + blockSize - 1) / blockSize;

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    MatrixVectorMul<<<nBlocks, blockSize>>>(size, size, pitch, d_a, d_v, d_res);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipDeviceSynchronize();
    
    float run_time_ms = 0;
    hipEventElapsedTime(&run_time_ms, start, stop);
    std::cout << run_time_ms << std::endl;
    
    hipMemcpy(res, d_res, size * sizeof(float), hipMemcpyDeviceToHost);
    
    free(a);
    free(v);
    free(res);
    hipFree(d_a);
    hipFree(d_v);
    hipFree(d_res);
    return 0;
}
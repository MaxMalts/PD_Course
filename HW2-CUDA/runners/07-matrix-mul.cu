#include "hip/hip_runtime.h"
#include <iostream>
#include <MatrixMul.cuh>

int main() {
    int nElements = 0, blockSize = 0;
    std::cin >> nElements >> blockSize;
    int size = static_cast<int>(sqrt(nElements));
    
    float* a = (float*)malloc(size * size * sizeof(float));
    float* b = (float*)malloc(size * size * sizeof(float));
    float* res = (float*)malloc(size * size * sizeof(float));
    
    float *d_a, *d_b, *d_res;
    hipMalloc(&d_a, size * size * sizeof(float));
    hipMalloc(&d_b, size * size * sizeof(float));
    hipMalloc(&d_res, size * size * sizeof(float));

    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            a[i * size + j] = 2.0f;
            b[i * size + j] = 3.0f;
        }
    }
    
    hipMemcpy(d_a, a, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * size * sizeof(float), hipMemcpyHostToDevice);

    dim3 nBlocks((size + blockSize - 1) / blockSize, (size + blockSize - 1) / blockSize);
    dim3 blockSizes(sqrt(blockSize), sqrt(blockSize));

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    MatrixMul<<<nBlocks, blockSizes>>>(size, size, size, d_a, d_b, d_res);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipDeviceSynchronize();
    hipGetLastError();
    
    float run_time_ms = 0;
    hipEventElapsedTime(&run_time_ms, start, stop);
    std::cout << run_time_ms << std::endl;
    
    hipMemcpy(res, d_res, size * size * sizeof(float), hipMemcpyDeviceToHost);
    
    free(a);
    free(b);
    free(res);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
    return 0;
}